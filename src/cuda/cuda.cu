
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAX_N 512

struct Matrix
{
  int size;
  double mat[MAX_N][MAX_N];
};

struct FreqMatrix
{
  int size;
  hipDoubleComplex mat[MAX_N][MAX_N];
};

void readMatrix(struct Matrix *m)
{
  scanf("%d", &(m->size));
  for (int i = 0; i < m->size; i++)
    for (int j = 0; j < m->size; j++)
      scanf("%lf", &(m->mat[i][j]));
}

__global__ void computeDFT(struct Matrix *src, struct FreqMatrix *dest)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < src->size && j < src->size)
  {
    hipDoubleComplex element = make_hipDoubleComplex(0.0, 0.0);
    for (int m = 0; m < src->size; m++)
    {
      for (int n = 0; n < src->size; n++)
      {
        double arg = (i * m / (double)src->size) + (j * n / (double)src->size);
        hipDoubleComplex exponent = make_hipDoubleComplex(cos(-2.0 * M_PI * arg), sin(-2.0 * M_PI * arg));
        element = hipCadd(element, hipCmul(make_hipDoubleComplex(src->mat[m][n], 0.0), exponent));
      }
    }
    dest->mat[i][j] = make_hipDoubleComplex(0.0, 0.0);
    dest->mat[i][j] = hipCdiv(element, make_hipDoubleComplex(src->size * src->size, 0.0));
  }
}

int main(void)
{
  struct Matrix source;
  struct FreqMatrix freq_domain;
  clock_t start, end;
  readMatrix(&source);
  freq_domain.size = source.size;

  start = clock();
  int threads = 32;
  dim3 threadsPerBlock(threads, threads);
  dim3 blocksPerGrid(source.size / threads, source.size / threads);

  struct Matrix *dev_source;
  struct FreqMatrix *dev_dest;
  hipMalloc((void **)&dev_source, sizeof(struct Matrix));
  hipMalloc((void **)&dev_dest, sizeof(struct FreqMatrix));
  hipMemcpy(dev_source, &source, sizeof(struct Matrix), hipMemcpyHostToDevice);
  computeDFT<<<blocksPerGrid, threadsPerBlock>>>(dev_source, dev_dest);
  hipMemcpy(&freq_domain, dev_dest, sizeof(struct FreqMatrix), hipMemcpyDeviceToHost);
  hipFree(dev_source);
  hipFree(dev_dest);

  hipDeviceSynchronize();
  end = clock();

  hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);

  // print some of the matrix
  for (int k = 0; k < 3; k++)
  {
    printf("{");
    for (int l = 0; l < 3; l++)
    {
      printf("(%lf, %lf), ", hipCreal(freq_domain.mat[k][l]), hipCimag(freq_domain.mat[k][l]));
    }
    printf("}\n");
  }

  // calculate the sum of the matrix
  for (int k = 0; k < source.size; k++)
  {
    for (int l = 0; l < source.size; l++)
    {
      sum = hipCadd(sum, freq_domain.mat[k][l]);
    }
  }

  sum = hipCdiv(sum, make_hipDoubleComplex(source.size, 0.0));
  printf("Average : (%lf, %lf)\n", hipCreal(sum), hipCimag(sum));
  printf("Time: %f\n", ((double)(end - start)) / CLOCKS_PER_SEC);

  return 0;
}