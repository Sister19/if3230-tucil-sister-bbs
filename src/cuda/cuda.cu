#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAX_N 512

struct Matrix
{
  int size;
  double mat[MAX_N][MAX_N];
};

struct FreqMatrix
{
  int size;
  hipDoubleComplex mat[MAX_N][MAX_N];
};

void readMatrix(struct Matrix *m)
{
  scanf("%d", &(m->size));
  for (int i = 0; i < m->size; i++)
    for (int j = 0; j < m->size; j++)
      scanf("%lf", &(m->mat[i][j]));
}

hipDoubleComplex dft(struct Matrix *mat, int k, int l)
{
  hipDoubleComplex element = 0.0;
  for (int m = 0; m < mat->size; m++)
  {
    for (int n = 0; n < mat->size; n++)
    {
      hipDoubleComplex arg = (k * m / (double)mat->size) + (l * n / (double)mat->size);
      hipDoubleComplex exponent = cexp(-2.0I * M_PI * arg);
      element += mat->mat[m][n] * exponent;
    }
  }
  return element / (double)(mat->size * mat->size);
}

__global void computeMatrix(struct Matrix *src, struct Matrix *dest, int k, int l)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < src->size && j < src->size)
  {
    hipDoubleComplex element = 0.0;
    for (int m = 0; m < src->size; m++)
    {
      for (int n = 0; n < src->size; n++)
      {
        hipDoubleComplex arg = (k * m / (double)src->size) + (l * n / (double)src->size);
        hipDoubleComplex exponent = cexp(-2.0I * M_PI * arg);
        element += src->mat[m][n] * exponent;
      }
    }
    dest->mat[i][j] = element / (double)(src->size * src->size);
  }
}

int main(void)
{
  struct Matrix source;
  struct FreqMatrix freq_domain;
  clock_t start, end;
  readMatrix(&source);
  freq_domain.size = source.size;

  start = clock();
  int threadsPerBlock = 32;
  dim3 threadsPerBlock(threadsPerBlock, threadsPerBlock);
  dim3 blocksPerGrid(source.size / threadsPerBlock, source.size / threadsPerBlock);

  for (int k = 0; k < source.size; k++)
  {
    for (int l = 0; l < source.size; l++)
    {
      struct Matrix *dev_source, *dev_dest;
      hipMalloc((void **)&dev_source, sizeof(struct Matrix));
      hipMalloc((void **)&dev_dest, sizeof(struct Matrix));
      hipMemcpy(dev_source, &source, sizeof(struct Matrix), hipMemcpyHostToDevice);
      computeMatrix<<<blocksPerGrid, threadsPerBlock>>>(dev_source, dev_dest, k, l);
      hipMemcpy(&freq_domain.mat[k][l], &dev_dest->mat[k][l], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
      hipFree(dev_source);
      hipFree(dev_dest);
    }
  }
  hipDeviceSynchronize();
  end = clock();

  hipDoubleComplex sum = 0.0;
  for (int k = 0; k < source.size; k++)
  {
    for (int l = 0; l < source.size; l++)
    {
      hipDoubleComplex el = freq_domain.mat[k][l];
      printf("(%lf, %lf) ", creal(el), cimag(el));
      sum += el;
    }
    printf("\n");
  }
  sum /= source.size;
  printf("Average : (%lf, %lf)\n", creal(sum), cimag(sum));
  printf("Time: %f\n", ((double)(end - start)) / CLOCKS_PER_SEC);

  return 0;
}