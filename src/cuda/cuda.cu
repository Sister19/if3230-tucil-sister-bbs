
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAX_N 512

struct Matrix
{
  int size;
  double mat[MAX_N][MAX_N];
};

struct FreqMatrix
{
  int size;
  hipDoubleComplex mat[MAX_N][MAX_N];
};

void readMatrix(struct Matrix *m)
{
  scanf("%d", &(m->size));
  for (int i = 0; i < m->size; i++)
    for (int j = 0; j < m->size; j++)
      scanf("%lf", &(m->mat[i][j]));
}

// cuDoubleComplex dft(struct Matrix *mat, int k, int l)
// {
//   double complex element = 0.0;
//   for (int m = 0; m < mat->size; m++)
//   {
//     for (int n = 0; n < mat->size; n++)
//     {
//       double complex arg = (k * m / (double)mat->size) + (l * n / (double)mat->size);
//       double complex exponent = cexp(-2.0I * M_PI * arg);
//       element += mat->mat[m][n] * exponent;
//     }
//   }
//   return element / (double)(mat->size * mat->size);
// }

__global__ void computeDFT(struct Matrix *src, struct FreqMatrix *dest)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < src->size && j < src->size)
  {
    hipDoubleComplex element = make_hipDoubleComplex(0.0, 0.0);
    for (int m = 0; m < src->size; m++)
    {
      for (int n = 0; n < src->size; n++)
      {
        // double complex arg = (k * m / (double)src->size) + (l * n / (double)src->size);
        // double complex exponent = cexp(-2.0I * M_PI * arg);
        double arg = (i * m / (double)src->size) + (j * n / (double)src->size);
        hipDoubleComplex exponent = make_hipDoubleComplex(cos(-2.0 * M_PI * arg), sin(-2.0 * M_PI * arg));
        element = hipCadd(element, hipCmul(make_hipDoubleComplex(src->mat[m][n], 0.0), exponent));
      }
    }
    dest->mat[i][j] = make_hipDoubleComplex(0.0, 0.0);
    dest->mat[i][j] = hipCdiv(element, make_hipDoubleComplex(src->size * src->size, 0.0));
  }
}

int main(void)
{
  struct Matrix source;
  struct FreqMatrix freq_domain;
  clock_t start, end;
  readMatrix(&source);
  freq_domain.size = source.size;

  start = clock();
  int threads = 32;
  dim3 threadsPerBlock(threads, threads);
  dim3 blocksPerGrid(source.size / threads, source.size / threads);

  struct Matrix *dev_source;
  struct FreqMatrix *dev_dest;
  hipMalloc((void **)&dev_source, sizeof(struct Matrix));
  hipMalloc((void **)&dev_dest, sizeof(struct FreqMatrix));
  hipMemcpy(dev_source, &source, sizeof(struct Matrix), hipMemcpyHostToDevice);
  computeDFT<<<blocksPerGrid, threadsPerBlock>>>(dev_source, dev_dest);
  hipMemcpy(&freq_domain, dev_dest, sizeof(struct FreqMatrix), hipMemcpyDeviceToHost);
  hipFree(dev_source);
  hipFree(dev_dest);

  hipDeviceSynchronize();
  end = clock();

  // double complex sum = 0.0
  hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
  for (int k = 0; k < source.size; k++)
  {
    for (int l = 0; l < source.size; l++)
    {
      // double complex el = freq_domain.mat[k][l];
      // printf("(%lf, %lf) ", creal(el), cimag(el));
      // sum += el;
      sum = hipCadd(sum, freq_domain.mat[k][l]);
      printf("(%lf, %lf) ", hipCreal(freq_domain.mat[k][l]), hipCimag(freq_domain.mat[k][l]));
    }
    printf("\n");
  }
  sum = hipCdiv(sum, make_hipDoubleComplex(source.size, 0.0));
  printf("Average : (%lf, %lf)\n", hipCreal(sum), hipCimag(sum));
  printf("Time: %f\n", ((double)(end - start)) / CLOCKS_PER_SEC);

  return 0;
}